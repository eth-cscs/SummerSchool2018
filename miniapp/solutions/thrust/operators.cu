#include "hip/hip_runtime.h"
//******************************************
// operators
// based on min-app code written by Oliver Fuhrer, MeteoSwiss
// modified by Ben Cumming, CSCS
// thrust implementation by Will Sawyer, CSCS
//
// implements
// *****************************************

// Description: Contains simple operators which can be used on 3d-meshes

#include "data.h"
#include "operators.h"
#include "stats.h"
#include <thrust/for_each.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>


namespace operators {

struct boundary_functor
{
    const int nx, ny;
    const double alpha, dxs;

    boundary_functor(int _nx, int _ny, double _alpha, double _dxs) : nx(_nx), ny(_ny), alpha(_alpha), dxs(_dxs) {}
    
    template <typename Tuple>   //     arguments: 0:count  1:BND_W  2:BND_E  3:BND_S  4:BND_N  5:X_OLD  6:U  7:S
    __host__ __device__
    void operator()(Tuple t)
    {
      int n = thrust::get<0>(t); 
      int i = n%nx;
      int j = n/nx;
      int nmi = n-i;
      int nmj = n-j;

      // On boundary but not on corner
      bool is_west  = (i==0) && (j>0) && (j<ny-1);
      bool is_east  = (i==nx-1) && (j>0) && (j<ny-1);
      bool is_south = (j==0) && (i>0) && (i<nx-1);
      bool is_north = (j==ny-1) && (i>0) && (i<nx-1);
      bool is_sw    = (i==0) && (j==0);
      bool is_nw    = (i==0) && (j==ny-1); 
      bool is_se    = (i==nx-1) && (j==0);
      bool is_ne    = (i==nx-1) && (j==ny-1);
      double my_val = thrust::get<6>(t);
      
      if(is_west) {
        thrust::get<7>(t) = -(4. + alpha) * my_val
                                                     + *(&thrust::get<6>(t)+1)
                          + *(&thrust::get<6>(t)+nx) + *(&thrust::get<6>(t)-nx)
                          + *(&thrust::get<1>(t)-nmj) // BND_W(j)
                          + alpha * thrust::get<5>(t)
                          + dxs * my_val * (1.0 - my_val);
      }
      if(is_east) {
        thrust::get<7>(t) = -(4. + alpha) * my_val
                          + *(&thrust::get<6>(t)-1) 
                          + *(&thrust::get<6>(t)+nx) + *(&thrust::get<6>(t)-nx)
                          + *(&thrust::get<2>(t)-nmj) // BND_E(j)
                          + alpha * thrust::get<5>(t)
                          + dxs * my_val * (1.0 - my_val);
      }
      if(is_south) {
        thrust::get<7>(t) = -(4. + alpha) * my_val
                          + *(&thrust::get<6>(t)-1) + *(&thrust::get<6>(t)+1)
                          + *(&thrust::get<6>(t)+nx) 
                          + *(&thrust::get<3>(t)-nmi) // BND_S(i)
                          + alpha * thrust::get<5>(t)
                          + dxs * my_val * (1.0 - my_val);
      }
      if(is_north) {
        thrust::get<7>(t) = -(4. + alpha) * my_val
                          + *(&thrust::get<6>(t)-1) + *(&thrust::get<6>(t)+1)
                                                    + *(&thrust::get<6>(t)-nx)
                          + *(&thrust::get<4>(t)-nmi) // BND_N(i)
                          + alpha * thrust::get<5>(t)
                          + dxs * my_val * (1.0 - my_val);
      }

// TODO:  implement the four corners:  SW, NW, SE, NE
      if(is_sw) {
        thrust::get<7>(t) = -(4. + alpha) * my_val
                                                     + *(&thrust::get<6>(t)+1)
                          + *(&thrust::get<6>(t)+nx)
                          + *(&thrust::get<1>(t)-nmj) // BND_W(j)
                          + *(&thrust::get<3>(t)-nmi) // BND_S(i)
                          + alpha * thrust::get<5>(t)
                          + dxs * my_val * (1.0 - my_val);
      }
      if(is_nw) {
        thrust::get<7>(t) = -(4. + alpha) * my_val
                                                     + *(&thrust::get<6>(t)+1)
                                                     + *(&thrust::get<6>(t)-nx)
                          + *(&thrust::get<1>(t)-nmj) // BND_W(j)
                          + *(&thrust::get<4>(t)-nmi) // BND_N(i)
                          + alpha * thrust::get<5>(t)
                          + dxs * my_val * (1.0 - my_val);
      }
      if(is_se) {
        thrust::get<7>(t) = -(4. + alpha) * my_val
                          + *(&thrust::get<6>(t)-1)
                          + *(&thrust::get<6>(t)+nx)
                          + *(&thrust::get<2>(t)-nmj) // BND_E(j)
                          + *(&thrust::get<3>(t)-nmi) // BND_S(i)
                          + alpha * thrust::get<5>(t)
                          + dxs * my_val * (1.0 - my_val);
      }
      if(is_ne) {
        thrust::get<7>(t) = -(4. + alpha) * my_val
                          + *(&thrust::get<6>(t)-1)
                                                     + *(&thrust::get<6>(t)-nx)
                          + *(&thrust::get<2>(t)-nmj) // BND_E(j)
                          + *(&thrust::get<4>(t)-nmi) // BND_N(i)
                          + alpha * thrust::get<5>(t)
                          + dxs * my_val * (1.0 - my_val);
      }
    }
};

struct interior_functor
{
    const int nx, ny;
    const double alpha, dxs;

    interior_functor(int _nx, int _ny, double _alpha, double _dxs) : nx(_nx), ny(_ny), alpha(_alpha), dxs(_dxs) {}
    
    template <typename Tuple>  // arguments: 0:count 1:X_OLD  2:U  3:S
    __host__ __device__
    void operator()(Tuple t)
    {
        int n = thrust::get<0>(t); // this is the counting iterator
        int i = n%nx;
        int j = n/nx;
        bool is_interior = i<(nx-1) && j<(ny-1) && i>0 && j>0;
        if(is_interior) {
           thrust::get<3>(t) = -(4. + alpha) * thrust::get<2>(t)          // central point
                                   + *(&thrust::get<2>(t)-1) + *(&thrust::get<2>(t)+1)    // east and west
                                   + *(&thrust::get<2>(t)-nx) + *(&thrust::get<2>(t)+nx)  // north and south
                                   + alpha * thrust::get<1>(t)
                                   + dxs * thrust::get<2>(t) * (1.0 - thrust::get<2>(t));
        }          
    }
};


void diffusion_thrust(int nx, int ny, double alpha, double dxs,
                      thrust::device_vector<double>& BND_W, thrust::device_vector<double>& BND_E,
                      thrust::device_vector<double>& BND_S, thrust::device_vector<double>& BND_N,
                      thrust::device_vector<double>& X_OLD, thrust::device_vector<double>& U, 
                      thrust::device_vector<double>& S
                     )
{
    const int N = nx*ny;
    thrust::counting_iterator<uint> n_first(0);
    thrust::counting_iterator<uint> n_last = n_first + N;

    // apply the transformation
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(n_first, X_OLD.begin(), U.begin(), S.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(n_last, X_OLD.end(),   U.end(), S.end())),
                     interior_functor(nx,ny,alpha,dxs));

// TODO:  zip up the tuple for the boundary_functor and invoke with for_each
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(n_first, BND_W.begin(), BND_E.begin(), BND_S.begin(), BND_N.begin(), X_OLD.begin(), U.begin(), S.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(n_last, BND_W.end(), BND_E.end(), BND_S.end(), BND_N.end(), X_OLD.end(), U.end(), S.end())),
                     boundary_functor(nx,ny,alpha,dxs));

}


} // namespace operators

