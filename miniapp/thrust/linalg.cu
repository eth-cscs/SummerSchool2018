#include "hip/hip_runtime.h"
// linear algebra subroutines
// Ben Cumming @ CSCS
// Will Sawyer @ CSCS (Thrust implementation)

#include <iostream>

#include <cmath>
#include <cstdio>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/inner_product.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>

#include "linalg.h"
#include "operators.h"
#include "data.h"
#include "stats.h"

struct diff_functor
{
    __host__ __device__
        double operator()(const double& x, const double& y) const {
            return x - y;
        }
};

struct square_functor
{
    __host__ __device__
        double operator()(const double& x) const { 
            return x * x;
        }
};

struct axpy_functor
{
    const double a;

    axpy_functor(double _a) : a(_a) {}

    __host__ __device__
        double operator()(const double& x, const double& y) const {
            return a * x + y;
        }
};

struct scale_functor
{
    const double a;

    scale_functor(double _a) : a(_a) {}

    __host__ __device__
        double operator()(const double& x) const {
            return a * x ;
        }
};

struct scaled_diff_functor
{
    const double a;

    scaled_diff_functor(double _a) : a(_a) {}

    __host__ __device__
        double operator()(const double& x, const double& y) const {
            return a * (x - y);
        }
};

struct lcomb_functor
{
    const double a, b;

    lcomb_functor(double _a, double _b) : a(_a), b(_b) {}

    __host__ __device__
        double operator()(const double& x, const double& y) const {
// TODO:  program the return value
        }
};

struct add_scaled_diff_functor
{
    const double a;
    add_scaled_diff_functor(double _a) : a(_a) {}

     template <typename Tuple>
    __host__ __device__
        void operator()(Tuple t) const {
// TODO: program Y = X + a * (L - R); where the arguments of the tuple are O:X  1:L  2:R  3:Y
        }
};

namespace linalg {

bool cg_initialized = false;

int calculate_grid_dim(const int block_dim, int n) {
    return (n-1)/block_dim + 1;
}

using namespace operators;
using namespace stats;

////////////////////////////////////////////////////////////////////////////////
//  blas level 1 reductions
////////////////////////////////////////////////////////////////////////////////

// computes the inner product of x and y
// x and y are vectors

double dot_thrust(thrust::device_vector<double>& X, thrust::device_vector<double>& Y)
{
         return thrust::inner_product(X.begin(), X.end(), Y.begin(), 0.0);
}

// computes the 2-norm of x
// x is a vector
double norm2_thrust(thrust::device_vector<double>& X)
{
// TODO: implement the norm using sqrt and thrust::inner_product
}

////////////////////////////////////////////////////////////////////////////////
//  blas level 1 vector-vector operations
////////////////////////////////////////////////////////////////////////////////

// computes y = x + alpha*(l-r)
// y, x, l and r are vectors
// alpha is a scalar
void add_scaled_diff_thrust(double A, thrust::device_vector<double>& X, thrust::device_vector<double>& L,
                            thrust::device_vector<double>& R, thrust::device_vector<double>& Y)
{
// TODO:  make tuple using make_zip_iterator where T = (X,L,R,Y)
}

// copy one vector into another y := x
// x and y are vectors of length N
void copy_thrust(thrust::device_vector<double>& X, thrust::device_vector<double>& Y)
{
    Y=X;
}

// sets x := value
// x is a vector
// value is a scalar
void fill_thrust(double A, thrust::device_vector<double>& X)
{
    thrust::fill(X.begin(),X.end(),A);
}

// computes y := alpha*x + y
// x and y are vectors
// alpha is a scalar
void axpy_thrust(double A, thrust::device_vector<double>& X, thrust::device_vector<double>& Y)
{
    // Y <- A * X + Y
    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), axpy_functor(A));
}

// computes y = alpha*(l-r)
// y, l and r are vectors of length N
// alpha is a scalar
void scaled_diff_thrust(double A, thrust::device_vector<double>& X, thrust::device_vector<double>& Y, thrust::device_vector<double>& Z)
{
    // Z <- A * (X - Y)
    thrust::transform(X.begin(), X.end(), Y.begin(), Z.begin(), scaled_diff_functor(A));
}

// computes y := alpha*x
// alpha is scalar
// y and x are vectors
void scale_thrust(double A, thrust::device_vector<double>& X, thrust::device_vector<double>& Y)
{
    thrust::transform(X.begin(), X.end(), Y.begin(), scale_functor(A));
}

// computes linear combination of two vectors y := alpha*x + beta*z
// alpha and beta are scalar
// y, x and z are vectors
void lcomb_thrust(double A, double B, thrust::device_vector<double>& X, thrust::device_vector<double>& Y, thrust::device_vector<double>& Z)
{
    // Z <- A * X + B * Y)
    thrust::transform(X.begin(), X.end(), Y.begin(), Z.begin(), lcomb_functor(A,B));
}


// conjugate gradient solver
// solve the linear system A*x = b for x
// the matrix A is implicit in the objective function for the diffusion equation
// the value in x constitute the "first guess" at the solution
// x(N)
// ON ENTRY contains all parameters BND_X and X_OLD, and the initial guess for the solution,
// ON EXIT  contains the solution

void cg_thrust(thrust::device_vector<double>& BND_W, thrust::device_vector<double>& BND_E, 
               thrust::device_vector<double>& BND_S, thrust::device_vector<double>& BND_N, 
               thrust::device_vector<double>& X_OLD, thrust::device_vector<double>& X, thrust::device_vector<double>& B, 
               const int maxiters, const double tol, bool& success)
{
    // this is the dimension of the linear system that we are to solve
    int nx = data::options.nx;
    int ny = data::options.ny;
    double ALPHA = data::options.alpha;
    double dxs = 1000. * (data::options.dx * data::options.dx);

    int length = nx*ny;

    // epsilon value use for matrix-vector approximation
    double eps     = 1.e-8;
    double eps_inv = 1. / eps;

// initialize memory for temporary storage
    
    thrust::device_vector<double> V(length, 0.0);
    thrust::device_vector<double> R(length, 0.0);
    thrust::device_vector<double> P(length, 0.0);
    thrust::device_vector<double> Ap(length, 0.0);
    thrust::device_vector<double> Fx(length, 0.0);
    thrust::device_vector<double> Fxold(length, 0.0);
    thrust::device_vector<double> Xhold(X);

    // matrix vector multiplication is approximated with
    // A*v = 1/epsilon * ( F(x+epsilon*v) - F(x) )
    //     = 1/epsilon * ( F(x+epsilon*v) - Fxold )
    // we compute Fxold at startup
    // we have to keep x so that we can compute the F(x+exps*v)
    // diffusion_raw(X_dv_ptr, Fxold_dv_ptr);

    diffusion_thrust( nx, ny, ALPHA, dxs, BND_W, BND_E, BND_S, BND_N, X_OLD, X, Fxold );

    // v = x + epsilon*x
    scale_thrust( (1.0+eps), X, V );

    // Fx = F(v)
    diffusion_thrust( nx, ny, ALPHA, dxs, BND_W, BND_E, BND_S, BND_N, X_OLD, V, Fx );

    // r = b - A*x
    // where A*x = (Fx-Fxold)/eps
    add_scaled_diff_thrust( -eps_inv, B, Fx, Fxold, R );

    // p = r
    P = R;

    // rold = <r,r>
    double rold = dot_thrust(R, R);
    double rnew = rold;

    // check for convergence
    success = sqrt(rold) < tol;
    if (success) {
        return;
    }

    int iter;
    for(iter=0; iter<maxiters; iter++) {
        // Ap = A*p
        lcomb_thrust( 1.0, eps, Xhold, P, V );

        diffusion_thrust( nx, ny, ALPHA, dxs, BND_W, BND_E, BND_S, BND_N, X_OLD, V, Fx );

        //ss_scaled_diff(Ap, eps_inv, Fx, Fxold);
        scaled_diff_thrust( eps_inv, Fx, Fxold, Ap );

        // alpha = rold / p'*Ap
        double alpha = rold / dot_thrust(P, Ap);

        // x += alpha*p
        axpy_thrust(alpha, P, X);

        // r -= alpha*Ap
        axpy_thrust(-alpha, Ap, R );

        // find new norm
        rnew = dot_thrust(R,R);

        // test for convergence
        if (sqrt(rnew) < tol) {
            success = true;
            break;
        }

        // p = r + (rnew/rold) * p
        lcomb_thrust( 1.0, rnew / rold, R, P, P);

        rold = rnew;
    }
    stats::iters_cg += iter + 1;

    if (!success) {
        std::cerr << "ERROR: CG_thrust failed to converge after " << iter
                  << " iterations, with residual " << sqrt(rnew)
                  << std::endl;
    }
}

} // namespace linalg
