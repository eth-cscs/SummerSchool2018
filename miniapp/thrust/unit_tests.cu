#include "hip/hip_runtime.h"
#include "linalg.h"

template <typename F>
bool run_test(F f, const char* name) {
    auto success = f();
    printf("%-25s : ", name);
    if(!success) {
        printf("\033[1;31mfailed\033[0m\n");
        return false;
    }
    printf("\033[1;32mpassed\033[0m\n");
    return true;
} 
template <typename T>
bool check_value(T value, T expected, T tol) {
    if(std::fabs(value-expected)>tol) {
        std::cout << "  expected " << expected << " got " << value << std::endl;
        return false;
    }
    return true;
}

bool test_scaled_diff() {
    auto n = 5;
    thrust::device_vector<double> y(n,0.0);
    thrust::device_vector<double> l(n,7.0);
    thrust::device_vector<double> r(n,2.0);
    
    linalg::scaled_diff_thrust(2.0, l, r, y);
    thrust::host_vector<double> y_out(y);

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(y_out[i], 10.0, 1.e-13);
    }
    return status;
}

bool test_fill() {
    auto n = 5;
    thrust::device_vector<double> x(n,3.0);
    linalg::fill_thrust(2.0,x);
    thrust::host_vector<double> x_out(x);

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(x_out[i], 2.0, 1.e-13);
    }
    return status;
}

bool test_axpy() {
    auto n = 5;
    thrust::device_vector<double> x(n,3.0);
    thrust::device_vector<double> y(n,5.0);
    linalg::axpy_thrust(0.5, x, y);

    thrust::host_vector<double> y_out(y);

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(y_out[i], (0.5*3.0 + 5.0), 1.e-13);
    }
    return status;
}

bool test_add_scaled_diff() {
    auto n = 5;
    thrust::device_vector<double> y(n,0.0);
    thrust::device_vector<double> x(n,3.0);
    thrust::device_vector<double> l(n,7.0);
    thrust::device_vector<double> r(n,2.0);

    linalg::add_scaled_diff_thrust(1.5, x, l, r, y);
    thrust::host_vector<double> y_out(y);

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(y_out[i], 3. + 1.5 * (7. - 2.), 1.e-13);
    }
    return status;
}

bool test_scale() {
    auto n = 5;
    thrust::device_vector<double> y(n,0.0);
    thrust::device_vector<double> x(n,3.0);

    for(auto i=0; i<n; ++i) {
        x[i] = 3.0;
    }

    linalg::scale_thrust(0.5, x, y);
    thrust::host_vector<double> y_out(y);

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(y_out[i], 1.5, 1.e-13);
    }
    return status;
}

bool test_lcomb() {
    auto n = 5;
    thrust::device_vector<double> y(n,0.0);
    thrust::device_vector<double> x(n,3.0);
    thrust::device_vector<double> z(n,7.0);

    linalg::lcomb_thrust(0.5, 2.0, x, z, y);
    thrust::host_vector<double> y_out(y);

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(y_out[i], (0.5*3. + 2.*7.), 1.e-13);
    }
    return status;
}

bool test_copy() {
    auto n = 5;
    thrust::device_vector<double> y(n,0.0);
    thrust::device_vector<double> x(n,3.0);

    linalg::copy_thrust(x,y);
    thrust::host_vector<double> x_out(x);
    thrust::host_vector<double> y_out(y);

    bool status = true;
    for(auto i=0; i<n; ++i) {
        status = status && check_value(y_out[i], x_out[i], 1.e-13);
    }

    return status;
}

bool test_dot() {
    auto n = 5;
    thrust::device_vector<double> y(n,7.0);
    thrust::device_vector<double> x(n,3.0);

    auto result = linalg::dot_thrust(x, y);

    return check_value(result, n*3.*7., 1.e-13);
}

bool test_norm2() {
    auto n = 5;
    thrust::device_vector<double> x(n,2.0);

    auto result = linalg::norm2_thrust(x);

    return check_value(result, sqrt(2.0 * 2.0 * 5.0), 1.e-13);
}

////////////////////////////////////////////////////////////////////////////////
// main
////////////////////////////////////////////////////////////////////////////////
int main(void) {
    run_test(test_scaled_diff,  "scaled_diff_thrust");
    run_test(test_fill,         "fill_thrust");
    run_test(test_axpy,         "axpy_thrust");
    run_test(test_add_scaled_diff, "add_scaled_diff_thrust");
    run_test(test_scale,        "scale_thrust");
    run_test(test_lcomb,        "lcomb_thrust");
    run_test(test_copy,         "copy_thrust");
    run_test(test_dot,          "dot_thrust");
    run_test(test_norm2,        "norm2_thrust");
}

